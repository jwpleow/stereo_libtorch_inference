#include "hip/hip_runtime.h"
#include "dcn_v2_im2col_cuda.h"
#include "dcn_v2.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <torch/script.h>
#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>

//extern THCState *state;

//THCState *state;

// THCState *state = at::globalContext().thc_state;

THCState *state = at::globalContext().lazyInitCUDA();

//THCState *state = at::globalContext().getTHCState();

__global__ void createBatchGemmBuffer(const float **input_b, float **output_b,
                                      float **columns_b, const float **ones_b,
                                      const float **weight_b, const float **bias_b,
                                      float *input, float *output,
                                      float *columns, float *ones,
                                      float *weight, float *bias,
                                      const int input_stride, const int output_stride,
                                      const int columns_stride, const int ones_stride,
                                      const int num_batches)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_batches)
    {
        input_b[idx] = input + idx * input_stride;
        output_b[idx] = output + idx * output_stride;
        columns_b[idx] = columns + idx * columns_stride;
        ones_b[idx] = ones + idx * ones_stride;
        // share weights and bias within a Mini-Batch
        weight_b[idx] = weight;
        bias_b[idx] = bias;
    }
}

at::Tensor
dcn_v2_cuda_forward(const at::Tensor &input,
                    const at::Tensor &weight,
                    const at::Tensor &bias,
                    const at::Tensor &offset,
                    const at::Tensor &mask,
                    const int64_t kernel_h,
                    const int64_t kernel_w,
                    const int64_t stride_h,
                    const int64_t stride_w,
                    const int64_t pad_h,
                    const int64_t pad_w,
                    const int64_t dilation_h,
                    const int64_t dilation_w,
                    const int64_t deformable_group)
{
    using scalar_t = float;
    //THCAssertSameGPU(THCudaTensor_checkGPU(state, 5, input, weight, bias, offset, mask));
    AT_ASSERTM(input.type().is_cuda(), "input must be a CUDA tensor");
    AT_ASSERTM(weight.type().is_cuda(), "weight must be a CUDA tensor");
    AT_ASSERTM(bias.type().is_cuda(), "bias must be a CUDA tensor");
    AT_ASSERTM(offset.type().is_cuda(), "offset must be a CUDA tensor");
    AT_ASSERTM(mask.type().is_cuda(), "mask must be a CUDA tensor");
    const int batch = input.size(0);
    const int channels = input.size(1);
    const int height = input.size(2);
    const int width = input.size(3);

    const int channels_out = weight.size(0);
    const int channels_kernel = weight.size(1);
    const int kernel_h_ = weight.size(2);
    const int kernel_w_ = weight.size(3);

    AT_ASSERTM(kernel_h_ == kernel_h && kernel_w_ == kernel_w,
               "Input shape and kernel shape wont match: (%d x %d vs %d x %d).", kernel_h_, kernel_w, kernel_h_, kernel_w_);

    AT_ASSERTM(channels == channels_kernel,
               "Input shape and kernel channels wont match: (%d vs %d).", channels, channels_kernel);

    const int height_out = (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
    const int width_out = (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;

    auto ones = at::ones({batch, height_out, width_out}, input.options());
    auto columns = at::empty({batch, channels * kernel_h * kernel_w, 1 * height_out * width_out}, input.options());
    auto output = at::empty({batch, channels_out, height_out, width_out}, input.options());

    int matrices_size = batch * sizeof(float *);

    auto input_b = static_cast<const float **>(THCudaMalloc(state, matrices_size));
    auto output_b = static_cast<float **>(THCudaMalloc(state, matrices_size));
    auto columns_b = static_cast<float **>(THCudaMalloc(state, matrices_size));
    auto ones_b = static_cast<const float **>(THCudaMalloc(state, matrices_size));
    auto weight_b = static_cast<const float **>(THCudaMalloc(state, matrices_size));
    auto bias_b = static_cast<const float **>(THCudaMalloc(state, matrices_size));

    const int block = 128;
    const int grid = (batch + block - 1) / block;

    createBatchGemmBuffer<<<grid, block, 0, at::cuda::getCurrentCUDAStream().stream()>>>(
        (const float**)input_b, output_b,
        columns_b, ones_b,
        weight_b, bias_b,
        input.data<scalar_t>(),
        output.data<scalar_t>(),
        columns.data<scalar_t>(),
        ones.data<scalar_t>(),
        weight.data<scalar_t>(),
        bias.data<scalar_t>(),
        channels * width * height,
        channels_out * width_out * height_out,
        channels * kernel_h * kernel_w * height_out * width_out,
        height_out * width_out,
        batch);

    long m_ = channels_out;
    long n_ = height_out * width_out;
    long k_ = 1;
    THCudaBlas_SgemmBatched(state,
                            't',
                            'n',
                            n_,
                            m_,
                            k_,
                            1.0f,
                            ones_b, k_,
                            bias_b, k_,
                            0.0f,
                            output_b, n_,
                            batch);

    modulated_deformable_im2col_cuda(at::cuda::getCurrentCUDAStream().stream(),
                                     input.data<scalar_t>(),
                                     offset.data<scalar_t>(),
                                     mask.data<scalar_t>(),
                                     batch, channels, height, width,
                                     height_out, width_out, kernel_h, kernel_w,
                                     pad_h, pad_w, stride_h, stride_w, dilation_h, dilation_w,
                                     deformable_group,
                                     columns.data<scalar_t>());

    long m = channels_out;
    long n = height_out * width_out;
    long k = channels * kernel_h * kernel_w;
    THCudaBlas_SgemmBatched(state,
                            'n',
                            'n',
                            n,
                            m,
                            k,
                            1.0f,
                            (const float **)columns_b, n,
                            weight_b, k,
                            1.0f,
                            output_b, n,
                            batch);

    THCudaFree(state, input_b);
    THCudaFree(state, output_b);
    THCudaFree(state, columns_b);
    THCudaFree(state, ones_b);
    THCudaFree(state, weight_b);
    THCudaFree(state, bias_b);
    return output;
}

